#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * Vector addition: C = A + B.
  *
  * This sample is a very basic sample that implements element by element
  * vector addition. It is the same as the sample illustrating Chapter 2
  * of the programming guide with some additions like error checking.
  */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""
#include "error_utils.h"
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */

double* vector_add_gpu(double* h_A, double* h_B, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(double);

    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host output vector C
    double* h_C = (double*)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector A
    double* d_A = NULL;
    err = hipMalloc((void**)&d_A, size);
    checkErr(err, "Allocating device vector A");

    // Allocate the device input vector B
    double* d_B = NULL;
    err = hipMalloc((void**)&d_B, size);
    checkErr(err, "Allocate device vector B");

    // Allocate the device output vector C
    double* d_C = NULL;
    err = hipMalloc((void**)&d_C, size);
    checkErr(err, "Allocate device vector C");

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Copy A host to device");

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Copy B host to device");

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Launch vectorAdd kernel");

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Copy C device to host");

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Free device vector C");

    printf("Done\n");
    return h_C;
}
